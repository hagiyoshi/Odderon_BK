#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h> 
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>
#include <cstring>
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <limits>
#include <cassert>
#include <cstdio>
#include <sstream>
#include <cmath>
#include <stdexcept>
#include <fstream>
#include <string>
#include <vector>
#include <cctype>
#include <complex>
#include <functional>

#include "Parameters.h"


__global__ void Integration_BK_direct(hipDoubleComplex* integrated, hipDoubleComplex* S_matrix,
	double* x_1, double* y_1, double h, int N_ini, int N_las, int N) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = j * N + i;
	if (i < N && j < N) {
		integrated[index] = make_hipDoubleComplex(0.0, 0.0);
		//sit the index which is center of the gaussian.

		hipDoubleComplex complex_zero = make_hipDoubleComplex(0.0, 0.0);
		//If x=N*j+i, then -x=N*(N-j)+N-i(when the origin is x= N*N/2 + N/2).
		for (int m = N_ini; m < N_las; m++) {
			for (int n = 0; n < N; n++) {
				double simpson1 = 1.0;
				double simpson2 = 1.0;
				int diffinitm = m - N_ini;
				if (m == N_ini || m == N_las - 1) {
					simpson1 = 1.0 / 3.0;
				}
				else if (diffinitm % 2 == 0) {
					simpson1 = 2.0 / 3.0;
				}
				else {

					simpson1 = 4.0 / 3.0;
				}


				if (n == 0 || n == N - 1) {
					simpson2 = 1.0 / 3.0;
				}
				else if (n % 2 == 0) {
					simpson2 = 2.0 / 3.0;
				}
				else {

					simpson2 = 4.0 / 3.0;
				}

				hipDoubleComplex trV_V = make_hipDoubleComplex(0.0, 0.0);
				//if r-z is out of the region then we take the S(r-z) =0.
				if((j - m + N / 2)<0|| (j - m + N / 2)>N-1|| (i - n + N / 2)<0|| (i - n + N / 2)>N-1){
					//trV= - S(r)
					trV_V = hipCsub(trV_V,
						S_matrix[j * N + i]);
				}
				else {
					//trV=S(r-z)
					trV_V = hipCadd(trV_V,
						S_matrix[(j - m + N / 2) * N + i - n + N / 2]);
					//trV=S(r-z)*S(-z) <- S(-x) = S(x)^*
					trV_V = hipCmul(trV_V,
						hipConj(S_matrix[m * N + n]));
					//trV=S(r-z)*S(-z) - S(r)
					trV_V = hipCsub(trV_V,
						S_matrix[j * N + i]);
				}

				hipDoubleComplex coeff = make_hipDoubleComplex(
					simpson1*simpson2
					*(x_1[j*N+i]* x_1[j*N + i]+ y_1[j*N + i] * y_1[j*N + i])
					/((x_1[m*N + n] - x_1[j*N + i])*(x_1[m*N + n] - x_1[j*N + i])+ (y_1[m*N + n] - y_1[j*N + i])*(y_1[m*N + n] - y_1[j*N + i]))
					/ (x_1[m*N + n] * x_1[m*N + n]+ y_1[m*N + n] * y_1[m*N+n]),
					0.0
				);

				if(((x_1[m*N + n] - x_1[j*N + i])*(x_1[m*N + n] - x_1[j*N + i]) 
						+ (y_1[m*N + n] - y_1[j*N + i])*(y_1[m*N + n] - y_1[j*N + i])) < 1.0e-12
					|| (x_1[m*N + n] * x_1[m*N + n] + y_1[m*N + n] * y_1[m*N+n]) < 1.0e-12){
					coeff = make_hipDoubleComplex(0.0,0.0);
				}

				integrated[index] = hipCadd(integrated[index], hipCmul(coeff, trV_V));

			}
		}

		hipDoubleComplex coeff2 = make_hipDoubleComplex(h*h*ALPHA_S_BAR/2.0/Pi, 0.0);

		integrated[index] = hipCmul(integrated[index], coeff2);
	}
}


void Integration_in_BK_equation(std::complex<double>* Smatrix_in, std::complex<double>* Integrated_out)
{


	int N = NX;
	double h = 1.0*LATTICE_SIZE / NX;
	double   xmax = h * NX / 1.0, xmin = -h * NX / 2.0, ymin = -h * NX / 2.0;
	double   *x = new double[N*N], *y = new double[N*N];
	for (int j = 0; j < NX; j++) {
		for (int i = 0; i < NX; i++)
		{
			x[NX*j + i] = xmin + i * h;
			y[NX*j + i] = ymin + j * h;
		}
	}

	// Allocate arrays on the device
	double  *x_d, *y_d;
	hipMalloc((void**)&x_d, sizeof(double)*N*N);
	hipMalloc((void**)&y_d, sizeof(double)*N*N);
	hipMemcpy(x_d, x, sizeof(double)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, sizeof(double)*N*N, hipMemcpyHostToDevice);

	hipDoubleComplex *S_matrix_d;
	hipMalloc((void**)&S_matrix_d, sizeof(hipDoubleComplex) * N*N);
	hipMemcpy(S_matrix_d, Smatrix_in, sizeof(std::complex<double>) * N*N, hipMemcpyHostToDevice);

	hipDoubleComplex *Integrated_d;
	hipMalloc((void**)&Integrated_d, sizeof(hipDoubleComplex)*N*N);

	dim3 dimGrid(int((N - 0.5) / BSZ) + 1, int((N - 0.5) / BSZ) + 1);
	dim3 dimBlock(BSZ, BSZ);
	
	Integration_BK_direct <<<dimGrid, dimBlock >>> (Integrated_d, S_matrix_d, x_d, y_d, h, 0, N, N);

	hipMemcpy(Integrated_out, Integrated_d, sizeof(std::complex<double>)*N*N, hipMemcpyDeviceToHost);


	hipFree(x_d);
	hipFree(y_d);;
	hipFree(Integrated_d);
	hipFree(S_matrix_d);
	delete[](x);
	delete[](y);
}